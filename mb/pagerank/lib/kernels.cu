#include "hip/hip_runtime.h"
#include <cmath>
#include <cassert>
#include <chrono>
#include "kernels.h"

#include "IO.hpp"
#include "Show.hpp"

/////////////////////////////////////////////////////////////////////////////
// GPU CODE 
/////////////////////////////////////////////////////////////////////////////

// texture<RefNumberType> texVec;
// texture<RefNumberType> texA;


inline unsigned PowerOfTwoAlign( unsigned int n )
{
	int PowerOfTwoAlign  = 1;
	while( PowerOfTwoAlign < n ) PowerOfTwoAlign = PowerOfTwoAlign << 1;
	return PowerOfTwoAlign;
} //*/


// KERNEL WITH BLOCKS. OK
__global__ void GPU_DENSE_PR(	RefNumberType* const dataA,
								const RefNumberType* const vecOld,
								RefNumberType* const vec,
								const double d )
{
	// row loop runs with blockIdx.x 
	const double InvFactor 	= 1/((double) gridDim.x );

	double sum = 0;
	for( size_t j = 0; j<gridDim.x; ++j )
	{
		// sum += A[row][j]*pold[j];
		sum += dataA[blockIdx.x*gridDim.x + j]*vecOld[j];
	}
	// sum now contains the scalar product A[row,:] DOT p
	vec[blockIdx.x] 	= d*sum + (1-d)*InvFactor;
}

// KERNEL WITH THREADS OK AND FASTER
__global__ void GPU_DENSE_PR_2(	RefNumberType* const dataA,
								const RefNumberType* const vecOld,
								RefNumberType* const vec,
								const double d, 
								int n )
{
	// loop runs with threads
	int tid = threadIdx.x + blockIdx.x*blockDim.x;
	const double InvFactor 	= 1/((double) n );

	if( tid < n )
	{

		double sum = 0;
		for( size_t j = 0; j<n; ++j )
		{
			// sum += A[row][j]*pold[j];
			sum += dataA[tid*n + j]*vecOld[j];
		}
		// sum now contains the scalar product A[row,:] DOT p
		vec[tid] 	= d*sum + (1-d)*InvFactor;
	}
} 

// KERNEL WITH THREADS + TEXTURE MEMORY
__global__ void GPU_DENSE_PR_3(	RefNumberType* const dataA,
								// const RefNumberType* const vecOld,
								RefNumberType* const vec,
								const double d,
								int n  )
{
	// row loop runs with blockIdx.x 
	int tid = threadIdx.x + blockIdx.x*blockDim.x;	
	const double InvFactor 	= 1/((double) n );
	
	if( tid < n )
	{
		double sum = 0;
		for( size_t j = 0; j<n; ++j )
		{
			// sum += A[row][j]*pold[j];
			// sum += dataA[threadIdx.x*blockDim.x + j]*vecOld[j];
			// sum += dataA[tid*n + j]*tex1Dfetch(texVec, (int) j);
			// sum += tex1Dfetch(texA, tid*n + j)*tex1Dfetch(texVec, j);
		}
		// sum now contains the scalar product A[row,:] DOT p
		vec[tid] 	= d*sum + (1-d)*InvFactor;
	}
} 

// KERNEL WITH THREADS
__global__ void GPU_CSR_PR(		const RefNumberType* const data,
								const size_t* const row_ptr,
								const size_t* const col_idx,
								const RefNumberType* const vecOld,
								RefNumberType* const vec,
								const double d,
								int n )
{
	// row loop runs with blockIdx.x 
	int tid = threadIdx.x + blockIdx.x*blockDim.x;	
	const double InvFactor 	= 1/((double) n );
	
	if( tid < n )
	{
		double sum = 0;

		for( size_t idx = row_ptr[tid]; idx < row_ptr[tid+1]; ++idx )
		{
			sum += data[idx]*vecOld[ col_idx[idx] ];
		}
		// sum now contains the scalar product A[row,:] DOT p
		vec[tid] 	= d*sum + (1-d)*InvFactor;
	}
} 

// KERNEL WITH THREADS
__global__ void GPU_CSR_PR_PartialSum(		const RefNumberType* const data,
											const size_t* const row_ptr,
											const size_t* const col_idx,
											const RefNumberType* const vecOld,
											const RefNumberType* const partialSum,
											RefNumberType* const vec,
											const double d,
											int n )
{
	// row loop runs with blockIdx.x 
	int tid = threadIdx.x + blockIdx.x*blockDim.x;	
	const double InvFactor 	= 1/((double) n );
	
	if( tid < n )
	{
		double sum = (*partialSum);

		for( size_t idx = row_ptr[tid]; idx < row_ptr[tid+1]; ++idx )
		{
			sum += data[idx]*vecOld[ col_idx[idx] ];
		}
		// sum now contains the scalar product A[row,:] DOT p
		vec[tid] 	= d*sum + (1-d)*InvFactor;
	}
} 

__global__ void GPU_Reduce_PartialSum( 	const RefNumberType* const vecOld,
									const size_t* const MaskLine,
									size_t MaskLine_size,
									RefNumberType* const result,
									RefNumberType defaultValue )
{
	(*result) = 0;
	for( size_t i = 0; i<MaskLine_size; ++i)
	{
		(*result) += vecOld[ MaskLine[i] ];
	}
	(*result) *= defaultValue;
}

__global__ void GPU_Reduce_Error(	RefNumberType* const tmpErr,
									RefNumberType* const vecOld,
									const RefNumberType* const vec, int n )
{
	(*tmpErr) = 0;
	for( size_t row = 0; row < n; ++row)
	{
		// on the fly compute norm( pold - pnext, 2) i.e. L2-norm between the current and last iteration
		(*tmpErr) 	   += (vec[row]-vecOld[row])*(vec[row]-vecOld[row]);
	}
}

const int THREADS_PER_BLOCK = 1024;
static_assert( THREADS_PER_BLOCK == 1024 , "Power of two value required!");
// int BlocksPerGrid = (n+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK;

__global__ void GPU_Reduce_Error_2(	RefNumberType* const partial_tmpErr,
									const RefNumberType* const vecOld,
									const RefNumberType* const vec, int n )
{
	//assert( THREADS_PER_BLOCK == PowerOfTwoAlign( THREADS_PER_BLOCK ) );
	__shared__ RefNumberType cache[THREADS_PER_BLOCK];

	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	float tmp = 0;
	while( tid < n )
	{
		tmp += (vec[tid]-vecOld[tid])*(vec[tid]-vecOld[tid]);
		tid += blockDim.x * gridDim.x;
	}

	cache[threadIdx.x] = tmp;

	__syncthreads();

	int i = blockDim.x / 2;

	while( i!= 0)
	{
		if( threadIdx.x  < i  )
		{
			cache[threadIdx.x] += cache[threadIdx.x + i];
		}
		__syncthreads();
		i/=2;
	}

	if( threadIdx.x == 0)
	{
		//(*tmpErr) = cache[0];
		partial_tmpErr[ blockIdx.x ] = cache[0];
	}
}


void showVec( RefNumberType* devPtr, int n )
{ 
	RefNumberType* tmpData = new RefNumberType[n];
	hipMemcpy( tmpData, devPtr, n*sizeof(RefNumberType), hipMemcpyDeviceToHost );
	for( int i = 0; i < n; ++ i )
	{
		printf("\t%.10e", (RefNumberType) tmpData[i]);
	}
	delete[] tmpData;
	printf("\n\n");
}

template<typename T> 
void showVecT( T* devPtr, int n )
{ 
	T* tmpData = new T[n];
	hipMemcpy( tmpData, devPtr, n*sizeof(T), hipMemcpyDeviceToHost );
	for( int i = 0; i < n; ++ i )
	{
		printf("\t%.10e", (T) tmpData[i]);
	}
	delete[] tmpData;
	printf("\n\n");
}

template<typename T> 
void showVecTu( T* devPtr, int n )
{ 
	T* tmpData = new T[n];
	hipMemcpy( tmpData, devPtr, n*sizeof(T), hipMemcpyDeviceToHost );
	for( int i = 0; i < n; ++ i )
	{
		printf("\t%u", (T) tmpData[i]);
	}
	delete[] tmpData;
	printf("\n\n");
}

std::vector<RefNumberType> GPU_PageRank_Dense( std::vector<std::vector<RefNumberType> > const &A, RefNumberType d, RefNumberType eps )
{	
	printf("GPU - CODE, n = %u \n", A.size() );

	//squared matrix required (n x n)
	assert( A.size() == A[0].size() );

	unsigned k 			= 0;
	size_t n 			= A.size();
	double InvFactor 	= 1/((double) n);
	RefNumberType tmpErr = 2*eps;
	int BlocksPerGrid = (n+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK;

	//------------------------------------------------------------------
	// GPU MEMORY
	//------------------------------------------------------------------
	RefNumberType *dev_dataA;
	hipMalloc( (void**) &dev_dataA, n*n*sizeof(RefNumberType) );

	RefNumberType *dev_vec;
	hipMalloc( (void**) &dev_vec, n*sizeof(RefNumberType) );

	RefNumberType *dev_vecOld;
	hipMalloc( (void**) &dev_vecOld, n*sizeof(RefNumberType) );

	RefNumberType *dev_partial_tmpErr;
	hipMalloc( (void**) &dev_partial_tmpErr, BlocksPerGrid*sizeof(RefNumberType) );

	// TEXTURE - MEMORY
	// hipBindTexture( NULL, texVec, dev_vecOld, n*sizeof(RefNumberType) );
	// hipBindTexture( NULL, texA, dev_dataA, n*n*sizeof(RefNumberType) );

	// hipEvent_t start, stop; 
	// hipEventCreate( &start );
	// hipEventCreate( &stop );


	//------------------------------------------------------------------
	// GPU UPLOAD
	//------------------------------------------------------------------
	RefNumberType* dataA = new RefNumberType[n*n];
	MatrixToFlatData( dataA, A );
	hipMemcpy( dev_dataA, dataA, n*n*sizeof(RefNumberType), hipMemcpyHostToDevice );
	delete[] dataA;

	// uniform vecotor of length n, with values 1/n at all positions
	RefNumberType* init_vec_data = new RefNumberType[n];
	for( int i = 0; i < n; ++ i )
	{
		init_vec_data[i] = InvFactor;
	}
	hipMemcpy( dev_vec, init_vec_data, n*sizeof(RefNumberType), hipMemcpyHostToDevice );
	delete[] init_vec_data;

	//float time_kernel = 0;
	//float time_kernel2 = 0;
	RefNumberType* partial_tmpErr = new RefNumberType[BlocksPerGrid];

	while( tmpErr > eps )
	{
		// compute p = dAp  + (1-d).*1/n.*[1 1 ... 1];
		// O(n^2): dense matrix-vector multiplication

		// printf("BEFORE\n");
		// showVec( dev_vecOld, n);
		// showVec( dev_vec, n);

		hipMemcpy( dev_vecOld, dev_vec, n*sizeof(RefNumberType), hipMemcpyDeviceToDevice );

		// printf("after copy\n");
		// showVec( dev_vecOld, n);
		// showVec( dev_vec, n);
		// hipEventRecord( start, 0 );

		// GPU_DENSE_PR<<<n,1>>>( dev_dataA, dev_vecOld, dev_vec, d );
		GPU_DENSE_PR_2<<< (n+31)/32, 32>>>( dev_dataA, dev_vecOld, dev_vec, d, n );
		// GPU_DENSE_PR_3<<< (n+31)/32, 32>>>( dev_dataA, /*dev_vecOld,*/ dev_vec, d, n );

		// hipEventRecord( stop, 0 );
		// hipEventSynchronize( stop );

		// float deltatime;
		// hipEventElapsedTime( &deltatime, start, stop );
		// time_kernel += deltatime;

		// printf("AFTER\n");
		// showVec( dev_vecOld, n);
		// showVec( dev_vec, n);
		// hipEventRecord( start, 0 );

		// printf("n = %i\n", n );
		// assert( n == THREADS_PER_BLOCK);
		// GPU_Reduce_Error<<<1,1>>>( dev_tmpErr, dev_vecOld, dev_vec, n);

		assert( THREADS_PER_BLOCK == PowerOfTwoAlign( THREADS_PER_BLOCK ) );
		GPU_Reduce_Error_2<<< BlocksPerGrid, THREADS_PER_BLOCK >>>( dev_partial_tmpErr, dev_vecOld, dev_vec, n);

		//------------------------------------------------------------------
		// GPU DOWNLOAD AND COMPLETE ERROR REDUCTION
		//------------------------------------------------------------------
		tmpErr = 0;
		hipMemcpy( partial_tmpErr, dev_partial_tmpErr, BlocksPerGrid*sizeof(RefNumberType), hipMemcpyDeviceToHost );

		for( unsigned i = 0; i<BlocksPerGrid; ++i)
		{
			tmpErr += partial_tmpErr[i];
		}

		// hipEventRecord( stop, 0 );
		// hipEventSynchronize( stop );
		// hipEventElapsedTime( &deltatime, start, stop );
		// time_kernel2 += deltatime;

		tmpErr = sqrt( tmpErr );
		printf("[k = %u]: %e\n", k++, tmpErr );

		// if( k > 1000 ) break;
	}
	delete[] partial_tmpErr;

	//------------------------------------------------------------------
	// GPU DOWNLOAD
	//------------------------------------------------------------------
	RefNumberType* tmpData = new RefNumberType[n];
	hipMemcpy( tmpData, dev_vec, n*sizeof(RefNumberType), hipMemcpyDeviceToHost );
	std::vector<RefNumberType> ret;
	ret.assign(tmpData, tmpData + n);
	delete[] tmpData;
	//------------------------------------------------------------------
	// GPU CLEANUP
	//------------------------------------------------------------------
	// hipEventDestroy( start );
	// hipEventDestroy( stop );

	// hipUnbindTexture( texVec );
	// hipUnbindTexture( texA );

	hipFree( dev_dataA );
	hipFree( dev_vec );
	hipFree( dev_vecOld );
	hipFree( dev_partial_tmpErr );

	return ret;
}


std::vector<RefNumberType> GPU_PageRank_CSR( CSRType<RefNumberType> const &S, size_t n, RefNumberType d, RefNumberType eps )
{	
	printf("GPU - CODE, nZZ = %u \n", S.data.size() );

	unsigned k 			 = 0;
	double InvFactor 	 = 1/((double) n);
	RefNumberType tmpErr = 2*eps;
	int BlocksPerGrid = (n+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK;

	//------------------------------------------------------------------
	// GPU MEMORY
	//------------------------------------------------------------------
	RefNumberType *dev_data;
	hipMalloc( (void**) &dev_data, S.data.size()*sizeof(RefNumberType) );

	size_t *dev_row_ptr;
	hipMalloc( (void**) &dev_row_ptr, S.row_ptr.size()*sizeof(size_t) );

	size_t *dev_col_idx;
	hipMalloc( (void**) &dev_col_idx, S.col_idx.size()*sizeof(size_t) );

	RefNumberType *dev_vec;
	hipMalloc( (void**) &dev_vec, n*sizeof(RefNumberType) );

	RefNumberType *dev_vecOld;
	hipMalloc( (void**) &dev_vecOld, n*sizeof(RefNumberType) );

	RefNumberType *dev_partial_tmpErr;
	hipMalloc( (void**) &dev_partial_tmpErr, BlocksPerGrid*sizeof(RefNumberType) );

	// TEXTURE - MEMORY
	// hipBindTexture( NULL, texVec, dev_vecOld, n*sizeof(RefNumberType) );
	// hipBindTexture( NULL, texA, dev_dataA, n*n*sizeof(RefNumberType) );

	//------------------------------------------------------------------
	// GPU UPLOAD
	//------------------------------------------------------------------
	RefNumberType* tmp1 = new RefNumberType[ S.data.size() ];
	std::copy( S.data.begin(), S.data.end(), tmp1 );
	hipMemcpy( dev_data, tmp1, S.data.size()*sizeof(RefNumberType), hipMemcpyHostToDevice );
	delete[] tmp1;

	size_t* tmp2 = new size_t[ S.row_ptr.size() ];
	std::copy( S.row_ptr.begin(), S.row_ptr.end(), tmp2 );
	hipMemcpy( dev_row_ptr, tmp2, S.row_ptr.size()*sizeof(size_t), hipMemcpyHostToDevice );
	delete[] tmp2;

	size_t* tmp3 = new size_t[ S.col_idx.size() ];
	std::copy( S.col_idx.begin(), S.col_idx.end(), tmp3 );
	hipMemcpy( dev_col_idx, tmp3, S.col_idx.size()*sizeof(size_t), hipMemcpyHostToDevice );
	delete[] tmp3;

	RefNumberType* init_vec_data = new RefNumberType[n];
	for( int i = 0; i < n; ++ i )
	{
		init_vec_data[i] = InvFactor;
	}
	hipMemcpy( dev_vec, init_vec_data, n*sizeof(RefNumberType), hipMemcpyHostToDevice );
	delete[] init_vec_data;

	// float time_kernel = 0;
	// float time_kernel2 = 0;
	RefNumberType* partial_tmpErr = new RefNumberType[BlocksPerGrid];

	while( tmpErr > eps )
	{

		// copy current state to old vector.
		hipMemcpy( dev_vecOld, dev_vec, n*sizeof(RefNumberType), hipMemcpyDeviceToDevice );

		// printf("after copy\n");
		// showVec( dev_vecOld, n);
		// showVec( dev_vec, n);

		// hipEventRecord( start, 0 );

		GPU_CSR_PR<<< (n+31)/32, 32>>>( dev_data, dev_row_ptr, dev_col_idx, dev_vecOld, dev_vec, d, n );

		// hipEventRecord( stop, 0 );
		// hipEventSynchronize( stop );

		// float deltatime;
		// hipEventElapsedTime( &deltatime, start, stop );
		// time_kernel += deltatime;

		// printf("AFTER\n");
		// showVec( dev_vecOld, n);
		// showVec( dev_vec, n);
		// hipEventRecord( start, 0 );

		// printf("n = %i\n", n );
		// assert( n == THREADS_PER_BLOCK);
		// GPU_Reduce_Error<<<1,1>>>( dev_tmpErr, dev_vecOld, dev_vec, n);
		assert( THREADS_PER_BLOCK == PowerOfTwoAlign( THREADS_PER_BLOCK ) );
		GPU_Reduce_Error_2<<< BlocksPerGrid, THREADS_PER_BLOCK >>>( dev_partial_tmpErr, dev_vecOld, dev_vec, n);

		//------------------------------------------------------------------
		// GPU DOWNLOAD AND COMPLETE ERROR REDUCTION
		//------------------------------------------------------------------
		tmpErr = 0;
		hipMemcpy( partial_tmpErr, dev_partial_tmpErr, BlocksPerGrid*sizeof(RefNumberType), hipMemcpyDeviceToHost );

		for( unsigned i = 0; i<BlocksPerGrid; ++i)
		{
			tmpErr += partial_tmpErr[i];
		}
		// hipEventRecord( stop, 0 );
		// hipEventSynchronize( stop );
		// hipEventElapsedTime( &deltatime, start, stop );
		// time_kernel2 += deltatime;

		tmpErr = sqrt( tmpErr );
		printf("[k = %u]: %e\n", k++, tmpErr );

		// if( k > 1000 ) break;
	}

	delete[] partial_tmpErr;

	//------------------------------------------------------------------
	// GPU DOWNLOAD
	//------------------------------------------------------------------
	RefNumberType* tmpData = new RefNumberType[n];
	hipMemcpy( tmpData, dev_vec, n*sizeof(RefNumberType), hipMemcpyDeviceToHost );
	std::vector<RefNumberType> ret;
	ret.assign(tmpData, tmpData + n);
	delete[] tmpData;
	//------------------------------------------------------------------
	// GPU CLEANUP
	//------------------------------------------------------------------
	// hipEventDestroy( start );
	// hipEventDestroy( stop );

	//hipUnbindTexture( texVec );
	//hipUnbindTexture( texA );

	hipFree( dev_data );
	hipFree( dev_row_ptr );
	hipFree( dev_col_idx );
	hipFree( dev_vec );
	hipFree( dev_vecOld );
	hipFree( dev_partial_tmpErr );

	return ret;
}

std::vector<RefNumberType> GPU_PageRank_CSR_OPT( CSRType<RefNumberType> const &S, size_t n,  std::vector<size_t> const &MaskLine, RefNumberType defaultValue, RefNumberType d, RefNumberType eps )
{
	printf("GPU - CODE, nZZ = %u \n", S.data.size() );

	unsigned k 			 = 0;
	double InvFactor 	 = 1/((double) n);
	RefNumberType tmpErr = 2*eps;
	int BlocksPerGrid = (n+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK;

	//------------------------------------------------------------------
	// GPU MEMORY
	//------------------------------------------------------------------
	RefNumberType *dev_data;
	hipMalloc( (void**) &dev_data, S.data.size()*sizeof(RefNumberType) );

	size_t *dev_row_ptr;
	hipMalloc( (void**) &dev_row_ptr, S.row_ptr.size()*sizeof(size_t) );

	size_t *dev_col_idx;
	hipMalloc( (void**) &dev_col_idx, S.col_idx.size()*sizeof(size_t) );

	size_t *dev_MaskLine;
	hipMalloc( (void**) &dev_MaskLine, MaskLine.size()*sizeof(size_t) );

	RefNumberType *dev_vec;
	hipMalloc( (void**) &dev_vec, n*sizeof(RefNumberType) );

	RefNumberType *dev_vecOld;
	hipMalloc( (void**) &dev_vecOld, n*sizeof(RefNumberType) );

	RefNumberType *dev_partial_tmpErr;
	hipMalloc( (void**) &dev_partial_tmpErr, BlocksPerGrid*sizeof(RefNumberType) );

	RefNumberType *dev_partialSum;
	hipMalloc( (void**) &dev_partialSum, sizeof(RefNumberType) );
	// TEXTURE - MEMORY
	// hipBindTexture( NULL, texVec, dev_vecOld, n*sizeof(RefNumberType) );
	// hipBindTexture( NULL, texA, dev_dataA, n*n*sizeof(RefNumberType) );

	//------------------------------------------------------------------
	// GPU UPLOAD
	//------------------------------------------------------------------
	RefNumberType* tmp1 = new RefNumberType[ S.data.size() ];
	std::copy( S.data.begin(), S.data.end(), tmp1 );
	hipMemcpy( dev_data, tmp1, S.data.size()*sizeof(RefNumberType), hipMemcpyHostToDevice );
	delete[] tmp1;

	size_t* tmp2 = new size_t[ S.row_ptr.size() ];
	std::copy( S.row_ptr.begin(), S.row_ptr.end(), tmp2 );
	hipMemcpy( dev_row_ptr, tmp2, S.row_ptr.size()*sizeof(size_t), hipMemcpyHostToDevice );
	delete[] tmp2;

	size_t* tmp3 = new size_t[ S.col_idx.size() ];
	std::copy( S.col_idx.begin(), S.col_idx.end(), tmp3 );
	hipMemcpy( dev_col_idx, tmp3, S.col_idx.size()*sizeof(size_t), hipMemcpyHostToDevice );
	delete[] tmp3;

	size_t* tmp4 = new size_t[ S.col_idx.size() ];
	std::copy( MaskLine.begin(), MaskLine.end(), tmp4 );
	hipMemcpy( dev_MaskLine, tmp4, MaskLine.size()*sizeof(size_t), hipMemcpyHostToDevice );
	delete[] tmp4;

	RefNumberType* init_vec_data = new RefNumberType[n];
	for( int i = 0; i < n; ++ i )
	{
		init_vec_data[i] = InvFactor;
	}
	hipMemcpy( dev_vec, init_vec_data, n*sizeof(RefNumberType), hipMemcpyHostToDevice );
	delete[] init_vec_data;

	RefNumberType* partial_tmpErr = new RefNumberType[BlocksPerGrid];

	while( tmpErr > eps )
	{

		// copy current state to old vector.
		hipMemcpy( dev_vecOld, dev_vec, n*sizeof(RefNumberType), hipMemcpyDeviceToDevice );

		GPU_Reduce_PartialSum<<<1,1>>>( dev_vecOld, dev_MaskLine, MaskLine.size(), dev_partialSum, defaultValue );

		GPU_CSR_PR_PartialSum<<< (n+31)/32, 32>>>( dev_data, dev_row_ptr, dev_col_idx, dev_vecOld, dev_partialSum, dev_vec, d, n );

		assert( THREADS_PER_BLOCK == PowerOfTwoAlign( THREADS_PER_BLOCK ) );
		GPU_Reduce_Error_2<<< BlocksPerGrid, THREADS_PER_BLOCK >>>( dev_partial_tmpErr, dev_vecOld, dev_vec, n);

		//------------------------------------------------------------------
		// GPU DOWNLOAD AND COMPLETE ERROR REDUCTION
		//------------------------------------------------------------------
		tmpErr = 0;
		hipMemcpy( partial_tmpErr, dev_partial_tmpErr, BlocksPerGrid*sizeof(RefNumberType), hipMemcpyDeviceToHost );

		for( unsigned i = 0; i<BlocksPerGrid; ++i)
		{
			tmpErr += partial_tmpErr[i];
		}
		tmpErr = sqrt( tmpErr );
		printf("[k = %u]: %e\n", k++, tmpErr );

		// if( k > 1000 ) break;
	}
	delete[] partial_tmpErr;

	//------------------------------------------------------------------
	// GPU DOWNLOAD
	//------------------------------------------------------------------
	RefNumberType* tmpData = new RefNumberType[n];
	hipMemcpy( tmpData, dev_vec, n*sizeof(RefNumberType), hipMemcpyDeviceToHost );
	std::vector<RefNumberType> ret;
	ret.assign(tmpData, tmpData + n);
	delete[] tmpData;
	//------------------------------------------------------------------
	// GPU CLEANUP
	//------------------------------------------------------------------
	hipFree( dev_data );
	hipFree( dev_row_ptr );
	hipFree( dev_col_idx );
	hipFree( dev_MaskLine );
	hipFree( dev_vec );
	hipFree( dev_vecOld );
	hipFree( dev_partial_tmpErr );
	hipFree( dev_partialSum );
	
	return ret;
}

//------------------------------------------------------------------
// THIS FUNCION IS ONLY USED FOR TESTABILITY WHILE DEVELOPMENT
//------------------------------------------------------------------
RefNumberType TestWrapper_KERNEL_ReduceError( RefNumberType* init_vec_1, RefNumberType* init_vec_2, int n, int THREADS)
{
	int BlocksPerGrid = (n+THREADS-1)/THREADS;
	//------------------------------------------------------------------
	// GPU MEMORY
	//------------------------------------------------------------------
	RefNumberType *dev_vec;
	hipMalloc( (void**) &dev_vec, n*sizeof(RefNumberType) );

	RefNumberType *dev_vecOld;
	hipMalloc( (void**) &dev_vecOld, n*sizeof(RefNumberType) );

	RefNumberType *dev_partial_tmpErr;
	hipMalloc( (void**) &dev_partial_tmpErr, BlocksPerGrid*sizeof(RefNumberType) );

	//------------------------------------------------------------------
	// GPU UPLOAD
	//------------------------------------------------------------------
	hipMemcpy( dev_vec, init_vec_1, n*sizeof(RefNumberType), hipMemcpyHostToDevice );
	hipMemcpy( dev_vecOld, init_vec_2, n*sizeof(RefNumberType), hipMemcpyHostToDevice );


	// GPU_Reduce_Error<<<1,1>>>( dev_tmpErr, dev_vecOld, dev_vec, n);
	assert( THREADS == PowerOfTwoAlign( THREADS ) );
	GPU_Reduce_Error_2<<< BlocksPerGrid, THREADS >>>( dev_partial_tmpErr, dev_vecOld, dev_vec, n);

	//------------------------------------------------------------------
	// GPU DOWNLOAD
	//------------------------------------------------------------------
	RefNumberType tmpErr = 0;
	RefNumberType* partial_tmpErr = new RefNumberType[BlocksPerGrid];
	hipMemcpy( partial_tmpErr, dev_partial_tmpErr, BlocksPerGrid*sizeof(RefNumberType), hipMemcpyDeviceToHost );

	for( unsigned i = 0; i<BlocksPerGrid; ++i)
	{
		tmpErr += partial_tmpErr[i];
	}

	delete[] partial_tmpErr;
	//------------------------------------------------------------------
	// GPU CLEANUP
	//------------------------------------------------------------------
	hipFree( dev_vec );
	hipFree( dev_vecOld );
	hipFree( dev_partial_tmpErr );

	return tmpErr;
}

