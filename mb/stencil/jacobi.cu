#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include "oprecomp.h"

// Grid boundary conditions
#define RIGHT 1.0
#define LEFT 1.0
#define TOP 1.0
#define BOTTOM 10.0

// precision
#ifdef SINGLE
  typedef float REAL;
#define TOLERANCE 0.0001f
#else
  typedef double REAL;
#define TOLERANCE 0.0001
#endif


// Algorithm settings
#define NPRINT 1000
#define MAX_ITER 200000


__global__
void stencil_sum(REAL*grid, REAL *grid_new, int nx, int ny)
{
  int index=blockIdx.x * blockDim.x +threadIdx.x; // global thread id

  int nrow=index/ny;
  int diff=index-(nrow*ny);
  int k=(nrow+1)*(ny+2)+diff+1;

  if (index<nx*ny) 
      grid_new[k]=0.25 * (grid[k-1]+grid[k+1] + grid[k-(ny+2)] + grid[k+(ny+2)]);
}

__global__
void stencil_norm(REAL*grid, REAL*arraynorm, int nx, int ny)
{
  int index=blockIdx.x * blockDim.x +threadIdx.x; // globEl thread id
  
  int nrow=index/ny;
  int diff=index-(nrow*ny);
  int k=(nrow+1)*(ny+2)+diff+1;

  if (index<nx*ny)
     arraynorm[index]=(REAL)pow(grid[k]*4.0-grid[k-1]-grid[k+1] - grid[k-(ny+2)] - grid[k+(ny+2)], 2);

}

//   
//  Taken from CUDA document. Uses  Reduce v4. 
//  Partial sums performed for each block
//  

__global__
void reduce(REAL* g_idata, REAL *g_odata, int nx, int ny) {
extern __shared__ REAL sdata[];

  int tid=threadIdx.x;
  int i=blockIdx.x*(blockDim.x*2) + threadIdx.x;

  if ( (i+blockDim.x) < (nx*ny) ) 
     sdata[tid]=g_idata[i]+g_idata[i+blockDim.x];
  else
     sdata[tid]=0.0;

  __syncthreads();

  for(int s=blockDim.x/2;s>0;s>>=1) {
     if (tid<s) {
        sdata[tid] += sdata[tid+s];
     }
     __syncthreads();
  }
  if (tid ==0) { 
      g_odata[blockIdx.x]=sdata[0];
  }
}

void getDeviceInfo() {

  int nDevices;
  hipGetDeviceCount(&nDevices);
  for (int i = 0; i < nDevices; i++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printf("Device Number: %d\n", i);
    printf("  Device name: %s\n", prop.name);
    printf("  Memory Clock Rate (KHz): %d\n",
           prop.memoryClockRate);
    printf("  Memory Bus Width (bits): %d\n",
           prop.memoryBusWidth);
    printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
           2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
  }

}


// MAIN LOOP 
int main(int argc, char*argv[]) {

  int k;
  REAL tmpnorm,bnorm,norm;

  printf("Jacobi 4-point stencil\n");
  printf("----------------------\n\n");

  if (argc !=3) {
    printf("usage: $argv[0] GRIDX GRIDY \n");
      return(1);
  }

  // GPU info
  getDeviceInfo();

 // One device
  hipSetDevice(0);

#ifdef SINGLE
  printf("Using single precision\n");
#else
  printf("Using double precision \n");
#endif
  printf("sizeof(REAL)=%d\n",sizeof(REAL));

  int nx=atoi(argv[1]);
  int ny=atoi(argv[2]);

  printf("grid size %d X %d \n",ny,ny);

// GPU threads/block

  int blockSize=256;
  int numBlocks = ((nx*ny)+blockSize-1)/blockSize;
  printf("numBlocks=%d\n",numBlocks);

//
// host allocated memory
//

  REAL *grid= (REAL*)malloc(sizeof(REAL)*(nx+2)*(ny+2));
  REAL *grid_new= (REAL*)malloc(sizeof(REAL)*(nx+2)*(ny+2));
  REAL *arraynorm= (REAL*)malloc(sizeof(REAL)*nx*ny);
  REAL*blocknorm=(REAL*)malloc(sizeof(REAL)*numBlocks);

  //
  // Device allocated memory
  //

  REAL *d_grid, *d_grid_new, *d_arraynorm, *d_blocknorm;
  hipMalloc(&d_grid,(nx+2)*(ny+2)*sizeof(REAL));
  hipMalloc(&d_grid_new,(nx+2)*(ny+2)*sizeof(REAL));
  hipMalloc(&d_arraynorm,nx*ny*sizeof(REAL));
  hipMalloc(&d_blocknorm,numBlocks*sizeof(REAL)); 

// shared memory size on GPU 
  int smemsize=blockSize*sizeof(REAL);

  // Initialise Grid boundaries
  int i,j;
  for (i=0;i<ny+2;i++) {
    grid_new[i]=grid[i]=TOP;
    j=(ny+2)*(nx+1)+i;
    grid_new[j]=grid[j]=BOTTOM;
  }
  for (i=1;i<nx+1;i++) {
    j=(ny+2)*i;
    grid_new[j]=grid[j]=LEFT;
    grid_new[j+ny+1]=grid[j+ny+1]=RIGHT;
  }
   
  // Initialise rest of grid
  for (i=1;i<=nx;i++) 
    for (j=1;j<=ny;j++)
      k=(ny+2)*i+j;
      grid_new[k]=grid[k]=0.0;
   
  // initial norm factor
  tmpnorm=0.0;
  for (i=1;i<=nx;i++) {
    for (j=1;j<=ny;j++) {
      k=(ny+2)*i+j;            
      tmpnorm=tmpnorm+(REAL)pow(grid[k]*4.0-grid[k-1]-grid[k+1] - grid[k-(ny+2)] - grid[k+(ny+2)], 2); 
    }
  }
  bnorm=sqrt(tmpnorm);

//  start oprecomp timing **
  oprecomp_start();

// copy arrays to device

  hipMemcpy(d_grid,grid,(nx+2)*(ny+2)*sizeof(REAL), hipMemcpyHostToDevice);
  hipMemcpy(d_grid_new,grid_new,(nx+2)*(ny+2)*sizeof(REAL), hipMemcpyHostToDevice);


//    MAIN LOOP 
  int iter;
  for (iter=0; iter<MAX_ITER; iter++) {

    // calculate norm array
    stencil_norm<<<numBlocks,blockSize>>>(d_grid,d_arraynorm,nx,ny); 
    
    // perform reduction
    reduce<<<numBlocks,blockSize,smemsize>>>(d_arraynorm,d_blocknorm,nx,ny);
    hipMemcpy(blocknorm,d_blocknorm,numBlocks*sizeof(REAL),hipMemcpyDeviceToHost);
 
    // sum up temporary block sums
    tmpnorm=0.0;
    for (i=0;i<numBlocks;i++) {
      tmpnorm=tmpnorm+blocknorm[i];
    }
   
    norm=(REAL)sqrt(tmpnorm)/bnorm;

    if (norm < TOLERANCE) break;

    stencil_sum<<<numBlocks,blockSize>>>(d_grid,d_grid_new,nx,ny);

  // Wait for GPU to finish
  hipDeviceSynchronize();

    REAL *temp=d_grid_new;
    d_grid_new=d_grid;
    d_grid=temp;

    if (iter % NPRINT ==0) printf("Iteration =%d ,Relative norm=%e\n",iter,norm);
  }

  printf("Terminated on %d iterations, Relative Norm=%e \n", iter,norm);
  
//  for (i=0;i<=nx+1;i++) {
//    for (j=0;j<=ny+1;j++){
//     printf("->%lf ",grid[j+i*(ny+2)]);
//    }
//    printf("\n");
//  }

// stop oprecomp timing **
   oprecomp_stop();

  hipFree(d_grid);
  hipFree(d_grid_new);
  hipFree(d_arraynorm);

  free(grid);
  free(grid_new);
  free(arraynorm);

  return 0;
    

  }
